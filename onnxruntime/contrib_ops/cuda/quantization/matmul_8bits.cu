#include "hip/hip_runtime.h"
// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "matmul_nbits.cuh"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T>
__device__ __forceinline__ T WarpUniform(T value) {
  struct {
    union {
      T value;
      uint32_t asInt;
    };
  } p;
  p.value = value;
  p.asInt = WARP_SHFL((unsigned)p.asInt, 0);
  return p.value;
}

__device__ __forceinline__ void AccumulateEightElements8b(uint64_t values_quant, half scale, uint8_t zp, const half* a, half* sums) {
  half2 scale_half2 = {scale, scale};
  half zp_adjust = -scale * __ushort2half_rn(zp);
  half2 zp_adjust2 = {zp_adjust, zp_adjust};
  uint4 vec_a = *(reinterpret_cast<const uint4*>(a));

  // Extract 8 uint8_t values from the 64-bit input.
  uint8_t q[8];
#pragma unroll
  for (int i = 0; i < 8; ++i) {
    q[i] = (values_quant >> (i * 8)) & 0xFF;
  }

  // Convert pairs to half2 (0,4 1,5 2,6 3,7 interleaved order).
  half2 element04 = __halves2half2(__ushort2half_rn(q[0]), __ushort2half_rn(q[4]));
  half2 element15 = __halves2half2(__ushort2half_rn(q[1]), __ushort2half_rn(q[5]));
  half2 element26 = __halves2half2(__ushort2half_rn(q[2]), __ushort2half_rn(q[6]));
  half2 element37 = __halves2half2(__ushort2half_rn(q[3]), __ushort2half_rn(q[7]));

  half2 v0 = element04 * scale_half2 + zp_adjust2;
  half2 v1 = element15 * scale_half2 + zp_adjust2;
  half2 v2 = element26 * scale_half2 + zp_adjust2;
  half2 v3 = element37 * scale_half2 + zp_adjust2;

  half2* sums_half2 = reinterpret_cast<half2*>(sums);
  sums_half2[0] = sums_half2[0] + v0 * (*(reinterpret_cast<half2*>(&(vec_a.x))));
  sums_half2[1] = sums_half2[1] + v1 * (*(reinterpret_cast<half2*>(&(vec_a.y))));
  sums_half2[2] = sums_half2[2] + v2 * (*(reinterpret_cast<half2*>(&(vec_a.z))));
  sums_half2[3] = sums_half2[3] + v3 * (*(reinterpret_cast<half2*>(&(vec_a.w))));
}

__device__ __forceinline__ void AccumulateEightElements8b(uint64_t values_quant, float scale, uint8_t zp, const float* a, float* sums) {
  float4 a_vec_0 = *(reinterpret_cast<const float4*>(a));
  float4 a_vec_1 = *(reinterpret_cast<const float4*>(a + 4));

  float zp_adjust = -scale * zp;
  float v0 = float(values_quant & 0xFF) * scale + zp_adjust;
  float v1 = float((values_quant >> 8) & 0xFF) * scale + zp_adjust;
  float v2 = float((values_quant >> 16) & 0xFF) * scale + zp_adjust;
  float v3 = float((values_quant >> 24) & 0xFF) * scale + zp_adjust;
  float v4 = float((values_quant >> 32) & 0xFF) * scale + zp_adjust;
  float v5 = float((values_quant >> 40) & 0xFF) * scale + zp_adjust;
  float v6 = float((values_quant >> 48) & 0xFF) * scale + zp_adjust;
  float v7 = float((values_quant >> 56) & 0xFF) * scale + zp_adjust;

  sums[0] += v0 * a_vec_0.x;
  sums[1] += v1 * a_vec_0.y;
  sums[2] += v2 * a_vec_0.z;
  sums[3] += v3 * a_vec_0.w;
  sums[4] += v4 * a_vec_1.x;
  sums[5] += v5 * a_vec_1.y;
  sums[6] += v6 * a_vec_1.z;
  sums[7] += v7 * a_vec_1.w;
}

constexpr int kColsPerThreadBlock = 8;
constexpr int kElementsPerThreadPerIteration = 8;
constexpr int kWarpSize = GPU_WARP_SIZE;

// kernel for 8bits quantized gemv, i.e., computing A(1,K) x B(K, N)
// B(K, N) is quantized blockwise with 8bits and stored as [N, (K + block_size - 1)/block_size, blob]
// The thread block size is (kWarpSize, kColsPerThreadBlock) and grid size is (N/kColsPerThreadBlock, 1)
// Each thread block computes [1, K] x [kColsPerThreadBlock, (K + block_size - 1)/block_size, blob],
//     i.e., computing kColsPerThreadBlock per block and a warp reduce (1, K) x (K)
template <class T, int block_size, bool has_zero_point>
__global__ void __launch_bounds__(kWarpSize* kColsPerThreadBlock) MatMulFloat8bKernel(
    T* output,
    const T* a_data,
    const uint8_t* b_data_quant,
    const T* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int blocks_per_K) {
  const int n_block_id = blockIdx.x;
  const int m_id = blockIdx.y;
  const int lane_id = threadIdx.x;
  const int warp_id = WarpUniform(threadIdx.y);
  const int n_id = n_block_id * kColsPerThreadBlock + warp_id;
  constexpr int k_per_iter = kWarpSize * kElementsPerThreadPerIteration;

  extern __shared__ char shared_buffer[];
  // load scale to shared buffer
  T* b_scale_vec = (T*)shared_buffer;
  int offset = n_block_id * kColsPerThreadBlock * blocks_per_K;
  for (int i = warp_id * kWarpSize + lane_id; i < kColsPerThreadBlock * blocks_per_K; i += kColsPerThreadBlock * kWarpSize) {
    b_scale_vec[i] = scales_data[offset + i];
  }

  uint8_t* b_zp_vec;
  (void)b_zp_vec;
  if constexpr (has_zero_point) {
    b_zp_vec = reinterpret_cast<uint8_t*>(b_scale_vec + kColsPerThreadBlock * blocks_per_K);
    const int b_zp_k = blocks_per_K;
    int zp_offset = n_block_id * kColsPerThreadBlock * b_zp_k;
    for (int i = warp_id * kWarpSize + lane_id; i < kColsPerThreadBlock * b_zp_k; i += kColsPerThreadBlock * kWarpSize) {
      b_zp_vec[i] = zero_points[zp_offset + i];
    }
    b_zp_vec += warp_id * b_zp_k;
  }
  __syncthreads();

  a_data += m_id * k + (lane_id << 3);

  b_scale_vec += warp_id * blocks_per_K;

  T sums[8] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f};
  int k_id = 0;
  int t_meta_k = lane_id * 8 / block_size;
  b_data_quant += n_id * blocks_per_K * (block_size / 2) + lane_id * 4;

#define UnRollReduction(unroll_size)                                                          \
  do {                                                                                        \
    constexpr int kUnroll = unroll_size;                                                      \
    constexpr int kUnrollMask = 0xffffffff & (~(kUnroll * k_per_iter - 1));                   \
    for (; k_id < (k & kUnrollMask); k_id += kUnroll * k_per_iter) {                          \
      _Pragma("unroll") for (int i = 0; i < kUnroll; i++) {                                   \
        uint64_t value = *(reinterpret_cast<const uint64_t*>(b_data_quant + k_per_iter * i)); \
        T scale = b_scale_vec[t_meta_k + k_per_iter / block_size * i];                        \
        uint8_t zp = 8;                                                                       \
        if constexpr (has_zero_point) {                                                       \
          zp = b_zp_vec[t_meta_k + k_per_iter / block_size * i];                              \
        }                                                                                     \
        AccumulateEightElements8b(value, scale, zp, a_data + k_id + i * k_per_iter, sums);    \
      }                                                                                       \
      b_data_quant += k_per_iter * kUnroll;                                                   \
      t_meta_k += k_per_iter / block_size * kUnroll;                                          \
    }                                                                                         \
  } while (false)

  UnRollReduction(16);
  UnRollReduction(4);
  UnRollReduction(1);
#undef UnRollReduction

  // handle reminder
  if (k_id + lane_id * 8 < k) {
    uint64_t value = *(reinterpret_cast<const uint64_t*>(b_data_quant));
    T scale = b_scale_vec[t_meta_k];
    uint8_t zp = 8;
    if constexpr (has_zero_point) {
      zp = b_zp_vec[t_meta_k];
    }
    AccumulateEightElements8b(value, scale, zp, a_data + k_id, sums);
  }

  float sum = (float)(sums[0] + sums[1] + sums[2] + sums[3] + sums[4] + sums[5] + sums[6] + sums[7]);
  // warp reduction
  for (int i = kWarpSize / 2; i > 0; i = i / 2) {
    sum += WARP_SHFL_DOWN(sum, i);
  }

  if (lane_id == 0) {
    output[m_id * n + n_id] = sum;
  }
}  // namespace cuda

template <class T>
bool TryMatMul8Bits(
    T* output,
    const T* a_data,
    const uint8_t* b_data_quant,
    const T* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int block_size,
    int shared_mem_per_block,
    hipStream_t stream) {
  if (n % kColsPerThreadBlock != 0 || k % 8 != 0 || m > 1) {
    return false;
  }
  dim3 blocks((n + kColsPerThreadBlock - 1) / kColsPerThreadBlock, m);
  dim3 threads(GPU_WARP_SIZE_HOST, kColsPerThreadBlock);
  int blocks_per_K = (k + block_size - 1) / block_size;
  int shared_mem_size = sizeof(T) * blocks_per_K * kColsPerThreadBlock +
                        (zero_points != nullptr ? (blocks_per_K + 1) / 2 * kColsPerThreadBlock * 2 : 0);
  if (shared_mem_size > shared_mem_per_block) {
    return false;
  }

#define MatMulFloat8bKernelDispatch(block_size)                                              \
  if (nullptr != zero_points) {                                                              \
    MatMulFloat8bKernel<T, block_size, true><<<blocks, threads, shared_mem_size, stream>>>(  \
        output, a_data, b_data_quant, scales_data, zero_points, m, n, k, blocks_per_K);      \
  } else {                                                                                   \
    MatMulFloat8bKernel<T, block_size, false><<<blocks, threads, shared_mem_size, stream>>>( \
        output, a_data, b_data_quant, scales_data, zero_points, m, n, k, blocks_per_K);      \
  }

  if (16 == block_size) {
    MatMulFloat8bKernelDispatch(16);
  } else if (32 == block_size) {
    MatMulFloat8bKernelDispatch(32);
  } else if (64 == block_size) {
    MatMulFloat8bKernelDispatch(64);
  } else if (128 == block_size) {
    MatMulFloat8bKernelDispatch(128);
  } else {
    ORT_THROW("block size ", block_size, " is not supported");
  }

#undef MatMulFloat8bKernelDispatch

  return true;
}

template bool TryMatMul8Bits<float>(
    float* output,
    const float* a_data,
    const uint8_t* b_data_quant,
    const float* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int block_size,
    int shared_mem_per_block,
    hipStream_t stream);

template bool TryMatMul8Bits<half>(
    half* output,
    const half* a_data,
    const uint8_t* b_data_quant,
    const half* scales_data,
    const uint8_t* zero_points,
    int m,
    int n,
    int k,
    int block_size,
    int shared_mem_per_block,
    hipStream_t stream);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
